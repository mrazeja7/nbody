#include "hip/hip_runtime.h"
/* nbody.cu */
/*
this is a version of the implementation that is supposed to be run on an NVIDIA GPU.
It's a rather straightforward design, with each body calculating its own force vector
from all other bodies in one thread. I think this is appropriate granularity-wise,
if I wanted to use a thread for every (i,j) pair I would potentially spawn
trillions of threads, which is not favorable.
*/
#include <iostream>
#include <fstream>
#include <cmath>
#include <cstdio>
#include <string>
#include <algorithm>
#include <chrono>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdlib>
#include <random>
using namespace std;
using namespace std::chrono;
__device__ __constant__ float G = 6.674e-11;
#define BLOCKSIZE 512
float *xPos;
float *yPos;
float *xVel;
float *yVel;
float *mass;
int n;
int currentTime;
int32_t gifW, gifH, gifDelay;
float timeInSeconds;
float gflops;
int getCurrentTime()
{
	return currentTime;
}
int getCount()
{
	return n;
}

__global__ void updateKernel(float *xPos, float *yPos, float *xVel, float *yVel, float *mass, int count)
{
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID >= count)
		return;
	int i = threadID;
	float G = 6.674e-11;

	float Fx = 0.0f; // 1
	float Fy = 0.0f; // 1
	for (unsigned int j = 0; j < count; j++)
	{
		if (j != i)
		{
			float dx = xPos[j] - xPos[i]; // 2
			float dy = yPos[j] - yPos[i]; // 2
			float r2 = dx*dx + dy*dy + 0.001f; // 5
			float invertedR2 = 1.0f / r2; // 2
			Fx += dx * invertedR2 * mass[j]; // 4
			Fy += dy * invertedR2 * mass[j]; // 4
		}
	}
	xVel[i] += G * Fx * mass[i]; // 4
	yVel[i] += G * Fy * mass[i]; // 4
	xPos[i] += xVel[i]; // 2
	yPos[i] += yVel[i]; // 2
}

void optimizedUpdate() // estimated FLOP counts in comments on each line
{
	hipError_t cudaStatus;
	int nblocks = (n + BLOCKSIZE - 1) / BLOCKSIZE;
	cout << "attempting to start kernel,  " << nblocks << " " << BLOCKSIZE << endl;
	updateKernel << <nblocks, BLOCKSIZE >> >(xPos, yPos, xVel, yVel, mass, n);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		cout << "update failed: " << hipGetErrorString(cudaStatus) << endl;

	cout << "kernel should be running " << endl;
	hipDeviceSynchronize();
	currentTime++;
}
void setGifProps(int w, int h, int d)
{
	gifW = w;
	gifH = h;
	gifDelay = d;
}

__global__ void initializeBodies(float *xPos, float *yPos, float *xVel, float *yVel, float *mass, int count, int w, int h, int seed)
{
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID >= count)
		return;
	hiprandState_t state;
	hiprand_init(seed, 0, 0, &state);

	if (threadID != count-1)
	{
		xPos[threadID] = hiprand_uniform(&state) * w - w / 2;
		yPos[threadID] = hiprand_uniform(&state) * h - h / 2;
		mass[threadID] = hiprand_uniform(&state) * 10000.0 + 10000.0;
		xVel[threadID] = yVel[threadID] = 0.0;
	}
	else
	{
		xPos[threadID] = 0.0;
		yPos[threadID] = 0.0;
		mass[threadID] = 100000.0;
		xVel[threadID] = yVel[threadID] = 0.0;
	}
}

void randomBodies_onDev_(int count) // initialize everything on the device using a different kernel
{
	hipError_t cudaStatus;
	hipMalloc((void**)&xPos, count * sizeof(float));
	hipMalloc((void**)&yPos, count * sizeof(float));
	hipMalloc((void**)&xVel, count * sizeof(float));
	hipMalloc((void**)&yVel, count * sizeof(float));
	hipMalloc((void**)&mass, count * sizeof(float));

	// 1 thread is enough
	int nblocks = (count + BLOCKSIZE - 1) / BLOCKSIZE;
	initializeBodies << <nblocks, BLOCKSIZE >> > (xPos, yPos, xVel, yVel, mass, count, gifW, gifH, unsigned(time(NULL)));
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		cout << "kernel failed: " << hipGetErrorString(cudaStatus) << endl;

	hipDeviceSynchronize();
	n = count;
}

void randomBodies_onDev(int count) // initialize everything on the device using a different kernel
{
	cout << "starting malloc" << endl;
	hipError_t cudaStatus;
	float *dummy;
	cudaStatus = hipMalloc((void**)&dummy, count * sizeof(float));
	cout << "dummy malloc: " << hipGetErrorString(cudaStatus) << endl;
	cudaStatus = hipMalloc((void**)&xPos, count * sizeof(float));
	cout << "malloc: " << hipGetErrorString(cudaStatus) << endl;
	cudaStatus = hipMalloc((void**)&yPos, count * sizeof(float));
	cout << "malloc: " << hipGetErrorString(cudaStatus) << endl;
	cudaStatus = hipMalloc((void**)&xVel, count * sizeof(float));
	cout << "malloc: " << hipGetErrorString(cudaStatus) << endl;
	cudaStatus = hipMalloc((void**)&yVel, count * sizeof(float));
	cout << "malloc: " << hipGetErrorString(cudaStatus) << endl;
	cudaStatus = hipMalloc((void**)&mass, count * sizeof(float));
	cout << "malloc: " << hipGetErrorString(cudaStatus) << endl;

	// 1 thread is enough
	int nblocks = (count + BLOCKSIZE - 1) / BLOCKSIZE;
	initializeBodies << <nblocks, BLOCKSIZE >> > (xPos, yPos, xVel, yVel, mass, count, gifW, gifH, unsigned(time(NULL)));
	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		cout << "kernel failed: " << hipGetErrorString(cudaStatus) << endl;

	hipDeviceSynchronize();
	n = count;
}

void randomBodies(int count) // initialize on the host, then copy over to device
{
	hipMalloc((void**)&xPos, count * sizeof(float));
	hipMalloc((void**)&yPos, count * sizeof(float));
	hipMalloc((void**)&xVel, count * sizeof(float));
	hipMalloc((void**)&yVel, count * sizeof(float));
	hipMalloc((void**)&mass, count * sizeof(float));

	default_random_engine generator;
	std::uniform_int_distribution<int> xpos(-gifW / 2, gifW / 2);
	std::uniform_int_distribution<int> ypos(-gifH / 2, gifH / 2);
	std::uniform_real_distribution<float> massgen(10000.0, 20000.0);

	float *xPos_h = new float[count];
	float *yPos_h = new float[count];
	float *xVel_h = new float[count];
	float *yVel_h = new float[count];
	float *mass_h = new float[count];

	for (int i = 0; i < count - 1; ++i)
	{
		xPos_h[i] = xpos(generator);
		yPos_h[i] = ypos(generator);
		mass_h[i] = massgen(generator);
		xVel_h[i] = yVel_h[i] = 0.0;
	}
	xPos_h[count - 1] = 0.0;
	yPos_h[count - 1] = 0.0;
	mass_h[count - 1] = 100000.0;
	xVel_h[count - 1] = yVel_h[count - 1] = 0.0;
	n = count;

	hipMemcpy(xPos, xPos_h, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(yPos, yPos_h, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(xVel, xVel_h, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(yVel, yVel_h, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(mass, mass_h, count * sizeof(float), hipMemcpyHostToDevice);

	delete[] xPos_h;
	delete[] yPos_h;
	delete[] xVel_h;
	delete[] yVel_h;
	delete[] mass_h;
}

void checkBodies()
{
	float *tmp = new float[100];
	hipMemcpy(tmp, xVel, 100 * sizeof(float), hipMemcpyDeviceToHost);

	for (size_t i = 0; i < 20; i++)
	{
		cout << tmp[i] << " ";
	}
	cout << endl;
}

void simulate_(int bodies, int iters)
{
	setGifProps(1024, 1024, 1);
	randomBodies_onDev(bodies);
	cout << "generated on dev" << endl;
	high_resolution_clock::time_point start = high_resolution_clock::now();
	for (int i = 0; i < iters; ++i)
		optimizedUpdate();

	float finish = duration_cast<duration<float>>(high_resolution_clock::now() - start).count();
	// (19*(n-1) + 14)*n*k total floating point operations == (19n-5)*n*k
	int appxFlops = (19 * getCount() - 5)*getCount()*iters;
	timeInSeconds = finish;
	gflops = 1e-9 * appxFlops / finish;
	cout << "sim done" << endl;
}

void simulate(int bodies, int iters)
{
	setGifProps(1024, 1024, 1);
	cout << "set gif" << endl;
	randomBodies_onDev(bodies);
	cout << "generated on dev" << endl;
	high_resolution_clock::time_point start = high_resolution_clock::now();
	for (int i = 0; i < iters; ++i)
		optimizedUpdate();
	cout << "optimized" << endl;

	float finish = duration_cast<duration<float>>(high_resolution_clock::now() - start).count();
	// (19*(n-1) + 14)*n*k total floating point operations == (19n-5)*n*k
	int appxFlops = (19 * getCount() - 5)*getCount()*iters;
	timeInSeconds = finish;
	gflops = 1e-9 * appxFlops / finish;
	cout << "sim done" << endl;
}

int main(int argc, char **argv)
{
	//cout << "Number of bodies: ";
	int b;
	cin >> b;
	//cout << "Number of iterations: ";
	int k;
	cin >> k;

	cout << "CUDA" << /*" threads\t"*/"\t";
	simulate(b, k);
	cout << getCount() << /*" bodies\t"*/"\t" << getCurrentTime() << /*" iterations\t"*/"\t" << timeInSeconds << /*" seconds\t"*/"\t" << gflops /*<< " GFlops/s." */ << endl;
	//cout << getCount() << " bodies\n" << getCurrentTime() << " iterations\n" << timeInSeconds << " seconds\n" << gflops << " GFlops/s." << endl;

	checkBodies();
	//system("PAUSE");
	return 0;
}