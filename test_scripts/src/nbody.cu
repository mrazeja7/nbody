#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

/* nbody.cu */
/*
	this is a version of the implementation that is supposed to be run on an NVIDIA GPU.
	It's a rather straightforward design, with each body calculating its own force vector
	from all other bodies in one thread. I think this is appropriate granularity-wise,
	if I wanted to use a thread for every (i,j) pair I would potentially spawn 
	trillions of threads, which is not favorable.
*/
#include <iostream>
#include <fstream>
#include <cmath>
#include <cstdio>
#include <string>
#include <algorithm>
#include <chrono>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdlib>
#include <random>
using namespace std;
using namespace std::chrono;
__device__ __constant__ float G = 6.674e-11;
#define BLOCKSIZE 512
float *xPos;
float *yPos;
float *xVel;
float *yVel;
float *mass;
uint64_t n;
uint64_t currentTime;
int32_t gifW, gifH, gifDelay;
float timeInSeconds;
float gflops;
uint64_t getCurrentTime()
{
	return currentTime;
}
uint64_t getCount()
{
	return n;
}

__global__ void updateKernel(float *xPos, float *yPos, float *xVel, float *yVel, float *mass, uint64_t count)
{
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID >= count)
		return;
	int i = threadID;
	float G = 6.674e-11;
	
	float Fx = 0.0f; // 1
	float Fy = 0.0f; // 1
	for (unsigned int j = 0; j < count; j++)
	{
		if (j != i)
		{
			float dx = xPos[j] - xPos[i]; // 2
			float dy = yPos[j] - yPos[i]; // 2
			float r2 = dx*dx + dy*dy + 0.001f; // 5
			float invertedR2 = 1.0f / r2; // 2
			Fx += dx * invertedR2 * mass[j]; // 4
			Fy += dy * invertedR2 * mass[j]; // 4
		}
	}
	xVel[i] += G * Fx * mass[i]; // 4
	yVel[i] += G * Fy * mass[i]; // 4
	xPos[i] += xVel[i]; // 2
	yPos[i] += yVel[i]; // 2
}

void optimizedUpdate() // estimated FLOP counts in comments on each line
{
	hipError_t cudaStatus;
	uint64_t nblocks = (n + BLOCKSIZE - 1) / BLOCKSIZE;
	updateKernel<<<nblocks, BLOCKSIZE>>>(xPos, yPos, xVel, yVel, mass, n);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		cout << "update failed: " << hipGetErrorString(cudaStatus) << endl;
	hipDeviceSynchronize();
	currentTime++;
}
void setGifProps(int w, int h, int d)
{
	gifW = w;
	gifH = h;
	gifDelay = d;
}

__global__ void initializeBodies(float *xPos, float *yPos, float *xVel, float *yVel, float *mass, uint64_t count, int w, int h, int seed)
{
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID >= 1)
		return;
	hiprandState_t state;
	hiprand_init(seed, 0, 0, &state);

	for (uint64_t i = 0; i < count - 1; ++i)
	{
		xPos[i] = hiprand_uniform(&state) * w - w / 2;
		yPos[i] = hiprand_uniform(&state) * h - h / 2;
		mass[i] = hiprand_uniform(&state) * 10000.0 + 10000.0;
		xVel[i] = yVel[i] = 0.0;
	}
	xPos[count - 1] = 0.0;
	yPos[count - 1] = 0.0;
	mass[count - 1] = 100000.0;
	xVel[count - 1] = yVel[count - 1] = 0.0;
}

void randomBodies_onDev(uint64_t count) // initialize everything on the device using a different kernel
{
	hipMalloc((void**)&xPos, count * sizeof(float));
	hipMalloc((void**)&yPos, count * sizeof(float));
	hipMalloc((void**)&xVel, count * sizeof(float));
	hipMalloc((void**)&yVel, count * sizeof(float));
	hipMalloc((void**)&mass, count * sizeof(float));

	// 1 thread is enough
	initializeBodies << <1, 32 >> > (xPos, yPos, xVel, yVel, mass, count, gifW, gifH, unsigned(time(NULL)));
	n = count;
}

void randomBodies(uint64_t count) // initialize on the host, then copy over to device
{
	hipMalloc((void**)&xPos, count * sizeof(float));
	hipMalloc((void**)&yPos, count * sizeof(float));
	hipMalloc((void**)&xVel, count * sizeof(float));
	hipMalloc((void**)&yVel, count * sizeof(float));
	hipMalloc((void**)&mass, count * sizeof(float));

	default_random_engine generator;
	std::uniform_int_distribution<int> xpos(-gifW / 2, gifW / 2);
	std::uniform_int_distribution<int> ypos(-gifH / 2, gifH / 2);
	std::uniform_real_distribution<float> massgen(10000.0, 20000.0);

	float *xPos_h = new float[count];
	float *yPos_h = new float[count];
	float *xVel_h = new float[count];
	float *yVel_h = new float[count];
	float *mass_h = new float[count];

	for (uint64_t i = 0; i < count - 1; ++i)
	{
		xPos_h[i] = xpos(generator);
		yPos_h[i] = ypos(generator);
		mass_h[i] = massgen(generator);
		xVel_h[i] = yVel_h[i] = 0.0;
	}
	xPos_h[count - 1] = 0.0;
	yPos_h[count - 1] = 0.0;
	mass_h[count - 1] = 100000.0;
	xVel_h[count - 1] = yVel_h[count - 1] = 0.0;
	n = count;
	
	hipMemcpy(xPos, xPos_h, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(yPos, yPos_h, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(xVel, xVel_h, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(yVel, yVel_h, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(mass, mass_h, count * sizeof(float), hipMemcpyHostToDevice);

	delete[] xPos_h;
	delete[] yPos_h;
	delete[] xVel_h;
	delete[] yVel_h;
	delete[] mass_h;
}

void simulate(int bodies, int iters)
{
	setGifProps(1024, 1024, 1);
	randomBodies(bodies);
	
	high_resolution_clock::time_point start = high_resolution_clock::now();
	for (int i = 0; i < iters; ++i)
		optimizedUpdate();

	float finish = duration_cast<duration<float>>(high_resolution_clock::now() - start).count();
	// (19*(n-1) + 14)*n*k total floating point operations == (19n-5)*n*k
	uint64_t appxFlops = (19 * getCount() - 5)*getCount()*iters;
	timeInSeconds = finish;
	gflops = 1e-9 * appxFlops / finish;
}

int main(int argc, char **argv)
{
	//cout << "Number of bodies: ";
	int b;
	cin >> b;
	//cout << "Number of iterations: ";
	int k;
	cin >> k;

	cout << "CUDA" << /*" threads\t"*/"\t";
	simulate(b, k);
	cout << getCount() << /*" bodies\t"*/"\t" << getCurrentTime() << /*" iterations\t"*/"\t" << timeInSeconds << /*" seconds\t"*/"\t" << gflops /*<< " GFlops/s." */<< endl;	
	//cout << getCount() << " bodies\n" << getCurrentTime() << " iterations\n" << timeInSeconds << " seconds\n" << gflops << " GFlops/s." << endl;	

	//system("PAUSE");
	return 0;
}
